#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

#include "../endian_utils.h"

typedef unsigned long long u64;
typedef unsigned int u32;
typedef unsigned char u8;
typedef u64 fe[4];

typedef struct pe {
  fe x, y, z;
} pe;

static_assert(IS_LITTLE_ENDIAN, "CUDA code requires little-endian");
static_assert(sizeof(fe)==32, "fe size");
static_assert(sizeof(pe)==96, "pe size");

#define INLINE __device__ __host__ inline __attribute__((always_inline))

__device__ __constant__ fe d_FE_P; // prime modulus loaded at runtime

static const fe FE_P_HOST = {0xfffffffefffffc2fULL, 0xffffffffffffffffULL,
                             0xffffffffffffffffULL, 0xffffffffffffffffULL};

INLINE u64 addc64(u64 x, u64 y, u64 ci, u64 *co) {
  unsigned __int128 t = (unsigned __int128)x + y + ci;
  *co = (u64)(t >> 64);
  return (u64)t;
}

INLINE u64 subc64(u64 x, u64 y, u64 ci, u64 *co) {
  u64 tmp = y + ci;
  *co = x < tmp;
  return x - tmp;
}

INLINE u64 umul128(u64 a, u64 b, u64 *hi) {
  unsigned __int128 t = (unsigned __int128)a * b;
  *hi = (u64)(t >> 64);
  return (u64)t;
}

INLINE void fe_clone(fe r, const fe a) { for(int i=0;i<4;i++) r[i]=a[i]; }
INLINE void fe_set64(fe r, u64 a) { r[0]=a; r[1]=r[2]=r[3]=0; }
INLINE int fe_cmp(const fe a, const fe b) {
  for(int i=3;i>=0;--i){ if(a[i]!=b[i]) return a[i]>b[i]?1:-1; }
  return 0;
}

INLINE void pe_clone(pe *r, const pe *a){
  // why: ensure struct copy works on host and device
  for(int i=0;i<4;i++){ r->x[i]=a->x[i]; r->y[i]=a->y[i]; r->z[i]=a->z[i]; }
}

INLINE void fe_mul_scalar(u64 r[5], const fe a, u64 b){
  u64 h1,h2,c=0; r[0]=umul128(a[0],b,&h1);
  r[1]=addc64(umul128(a[1],b,&h2),h1,c,&c);
  r[2]=addc64(umul128(a[2],b,&h1),h2,c,&c);
  r[3]=addc64(umul128(a[3],b,&h2),h1,c,&c);
  r[4]=addc64(0,h2,c,&c);
}

INLINE void fe_modp_add(fe r, const fe a, const fe b){
  u64 c=0; r[0]=addc64(a[0],b[0],c,&c); r[1]=addc64(a[1],b[1],c,&c);
  r[2]=addc64(a[2],b[2],c,&c); r[3]=addc64(a[3],b[3],c,&c);
  if(c){ r[0]=subc64(r[0],d_FE_P[0],0,&c); r[1]=subc64(r[1],d_FE_P[1],c,&c);
         r[2]=subc64(r[2],d_FE_P[2],c,&c); r[3]=subc64(r[3],d_FE_P[3],c,&c); }
}

INLINE void fe_modp_sub(fe r, const fe a, const fe b){
  u64 c=0; r[0]=subc64(a[0],b[0],c,&c); r[1]=subc64(a[1],b[1],c,&c);
  r[2]=subc64(a[2],b[2],c,&c); r[3]=subc64(a[3],b[3],c,&c);
  if(c){ r[0]=addc64(r[0],d_FE_P[0],0,&c); r[1]=addc64(r[1],d_FE_P[1],c,&c);
         r[2]=addc64(r[2],d_FE_P[2],c,&c); r[3]=addc64(r[3],d_FE_P[3],c,&c); }
}

INLINE void fe_modp_mul(fe r, const fe a, const fe b){
  u64 rr[8]={0},tt[5]={0},c=0;
  fe_mul_scalar(rr,a,b[0]);
  fe_mul_scalar(tt,a,b[1]);
  rr[1]=addc64(rr[1],tt[0],c,&c); rr[2]=addc64(rr[2],tt[1],c,&c);
  rr[3]=addc64(rr[3],tt[2],c,&c); rr[4]=addc64(rr[4],tt[3],c,&c);
  rr[5]=addc64(rr[5],tt[4],c,&c);
  fe_mul_scalar(tt,a,b[2]);
  rr[2]=addc64(rr[2],tt[0],c,&c); rr[3]=addc64(rr[3],tt[1],c,&c);
  rr[4]=addc64(rr[4],tt[2],c,&c); rr[5]=addc64(rr[5],tt[3],c,&c);
  rr[6]=addc64(rr[6],tt[4],c,&c);
  fe_mul_scalar(tt,a,b[3]);
  rr[3]=addc64(rr[3],tt[0],c,&c); rr[4]=addc64(rr[4],tt[1],c,&c);
  rr[5]=addc64(rr[5],tt[2],c,&c); rr[6]=addc64(rr[6],tt[3],c,&c);
  rr[7]=addc64(rr[7],tt[4],c,&c);
  fe_mul_scalar(tt,rr+4,0x1000003D1ULL);
  rr[0]=addc64(rr[0],tt[0],0,&c); rr[1]=addc64(rr[1],tt[1],c,&c);
  rr[2]=addc64(rr[2],tt[2],c,&c); rr[3]=addc64(rr[3],tt[3],c,&c);
  u64 hi,lo; lo=umul128(tt[4]+c,0x1000003D1ULL,&hi);
  r[0]=addc64(rr[0],lo,0,&c); r[1]=addc64(rr[1],hi,c,&c);
  r[2]=addc64(rr[2],0,c,&c); r[3]=addc64(rr[3],0,c,&c);
  if(fe_cmp(r,d_FE_P)>=0) fe_modp_sub(r,r,d_FE_P);
}

INLINE void fe_modp_sqr(fe r, const fe a){
  u64 rr[8]={0},tt[5]={0},c=0,t1,t2,lo,hi;
  rr[0]=umul128(a[0],a[0],&tt[1]);
  tt[3]=umul128(a[0],a[1],&tt[4]);
  tt[3]=addc64(tt[3],tt[3],0,&c); tt[4]=addc64(tt[4],tt[4],c,&c); t1=c;
  tt[3]=addc64(tt[1],tt[3],0,&c); tt[4]=addc64(tt[4],0,c,&c); t1+=c; rr[1]=tt[3];
  tt[0]=umul128(a[0],a[2],&tt[1]); tt[0]=addc64(tt[0],tt[0],0,&c);
  tt[1]=addc64(tt[1],tt[1],c,&c); t2=c; lo=umul128(a[1],a[1],&hi);
  tt[0]=addc64(tt[0],lo,0,&c); tt[1]=addc64(tt[1],hi,c,&c); t2+=c;
  tt[0]=addc64(tt[0],tt[4],0,&c); tt[1]=addc64(tt[1],t1,c,&c); t2+=c; rr[2]=tt[0];
  tt[3]=umul128(a[0],a[3],&tt[4]); lo=umul128(a[1],a[2],&hi);
  tt[3]=addc64(tt[3],lo,0,&c); tt[4]=addc64(tt[4],hi,c,&c); t1=c+c;
  tt[3]=addc64(tt[3],tt[3],0,&c); tt[4]=addc64(tt[4],tt[4],c,&c); t1+=c;
  tt[3]=addc64(tt[1],tt[3],0,&c); tt[4]=addc64(tt[4],t2,c,&c); t1+=c; rr[3]=tt[3];
  tt[0]=umul128(a[1],a[3],&tt[1]); tt[0]=addc64(tt[0],tt[0],0,&c);
  tt[1]=addc64(tt[1],tt[1],c,&c); t2=c; lo=umul128(a[2],a[2],&hi);
  tt[0]=addc64(tt[0],lo,0,&c); tt[1]=addc64(tt[1],hi,c,&c); t2+=c;
  tt[0]=addc64(tt[0],tt[4],0,&c); tt[1]=addc64(tt[1],t1,c,&c); t2+=c; rr[4]=tt[0];
  tt[3]=umul128(a[2],a[3],&tt[4]); tt[3]=addc64(tt[3],tt[3],0,&c);
  tt[4]=addc64(tt[4],tt[4],c,&c); t1=c; tt[3]=addc64(tt[3],tt[1],0,&c);
  tt[4]=addc64(tt[4],t2,c,&c); t1+=c; rr[5]=tt[3];
  tt[0]=umul128(a[3],a[3],&tt[1]); tt[0]=addc64(tt[0],tt[4],0,&c);
  tt[1]=addc64(tt[1],t1,c,&c); rr[6]=tt[0]; rr[7]=tt[1];
  fe_mul_scalar(tt,rr+4,0x1000003D1ULL);
  rr[0]=addc64(rr[0],tt[0],0,&c); rr[1]=addc64(rr[1],tt[1],c,&c);
  rr[2]=addc64(rr[2],tt[2],c,&c); rr[3]=addc64(rr[3],tt[3],c,&c);
  lo=umul128(tt[4]+c,0x1000003D1ULL,&hi);
  r[0]=addc64(rr[0],lo,0,&c); r[1]=addc64(rr[1],hi,c,&c);
  r[2]=addc64(rr[2],0,c,&c); r[3]=addc64(rr[3],0,c,&c);
  if(fe_cmp(r,d_FE_P)>=0) fe_modp_sub(r,r,d_FE_P);
}

INLINE void fe_modp_neg(fe r,const fe a){
  u64 c=0; r[0]=subc64(d_FE_P[0],a[0],c,&c);
  r[1]=subc64(d_FE_P[1],a[1],c,&c); r[2]=subc64(d_FE_P[2],a[2],c,&c);
  r[3]=subc64(d_FE_P[3],a[3],c,&c);
}

INLINE void _ec_jacobi_dbl1(pe *r,const pe *p){
  fe w,s,b,h,t; fe_modp_sqr(t,p->x); fe_modp_add(w,t,t);
  fe_modp_add(w,w,t); fe_modp_mul(s,p->y,p->z); fe_modp_mul(b,p->x,p->y);
  fe_modp_mul(b,b,s); fe_modp_add(b,b,b); fe_modp_add(b,b,b);
  fe_modp_add(t,b,b); fe_modp_sqr(h,w); fe_modp_sub(h,h,t);
  fe_modp_mul(r->x,h,s); fe_modp_add(r->x,r->x,r->x);
  fe_modp_sub(t,b,h); fe_modp_mul(t,w,t); fe_modp_sqr(r->y,p->y);
  fe_modp_sqr(h,s); fe_modp_mul(r->y,r->y,h); fe_modp_add(r->y,r->y,r->y);
  fe_modp_add(r->y,r->y,r->y); fe_modp_add(r->y,r->y,r->y);
  fe_modp_sub(r->y,t,r->y); fe_modp_mul(r->z,h,s); fe_modp_add(r->z,r->z,r->z);
  fe_modp_add(r->z,r->z,r->z); fe_modp_add(r->z,r->z,r->z);
}

INLINE void _ec_jacobi_add1(pe *r,const pe *p,const pe *q){
  fe u2,v2,u,v,w,a,vs,vc; fe_modp_mul(u2,p->y,q->z);
  fe_modp_mul(v2,p->x,q->z); fe_modp_mul(u,q->y,p->z);
  fe_modp_mul(v,q->x,p->z); assert(fe_cmp(v,v2)!=0);
  fe_modp_mul(w,p->z,q->z); fe_modp_sub(u,u,u2); fe_modp_sub(v,v,v2);
  fe_modp_sqr(vs,v); fe_modp_mul(vc,vs,v); fe_modp_mul(vs,vs,v2);
  fe_modp_mul(r->z,vc,w); fe_modp_sqr(a,u); fe_modp_mul(a,a,w);
  fe_modp_add(w,vs,vs); fe_modp_sub(a,a,vc); fe_modp_sub(a,a,w);
  fe_modp_mul(r->x,v,a); fe_modp_sub(a,vs,a); fe_modp_mul(a,a,u);
  fe_modp_mul(u,vc,u2); fe_modp_sub(r->y,a,u);
}

__global__ void mul_kernel(pe *r,const pe *p,const fe k,u32 bits){
  pe t; fe_clone(t.x,p->x); fe_clone(t.y,p->y); fe_clone(t.z,p->z);
  fe_set64(r->x,0); fe_set64(r->y,0); fe_set64(r->z,1);
  for(u32 i=0;i<bits;++i){
    if(k[i/64] & (1ULL<<(i%64))){
      if(r->x[0]==0 && r->y[0]==0) pe_clone(r,&t);
      else _ec_jacobi_add1(r,r,&t);
    }
    _ec_jacobi_dbl1(&t,&t);
  }
}

#define CUDA_CHECK_ERROR() \
  do { hipError_t e=hipGetLastError(); \
       if(e!=hipSuccess){fprintf(stderr,"CUDA %s:%d %s\n",__FILE__,__LINE__,hipGetErrorString(e));assert(0);} } while(0)

static void ensure_const(){
  static bool init=false; if(!init){
    hipMemcpyToSymbol(HIP_SYMBOL(d_FE_P),FE_P_HOST,sizeof(fe)); CUDA_CHECK_ERROR();
    init=true;
  }
}

extern "C" void ec_jacobi_mulrdc_cuda(pe *r,const pe *p,const fe k){
  ensure_const();
  pe *d_r,*d_p; fe *d_k; u32 bits=0; for(int i=3;i>=0;--i){ if(k[i]){ bits=64*i+ (64-__builtin_clzll(k[i])); break; }}
  hipMalloc(&d_r,sizeof(pe)); CUDA_CHECK_ERROR();
  hipMalloc(&d_p,sizeof(pe)); CUDA_CHECK_ERROR();
  hipMalloc(&d_k,sizeof(fe)); CUDA_CHECK_ERROR();
  hipMemcpy(d_p,p,sizeof(pe),hipMemcpyHostToDevice); CUDA_CHECK_ERROR();
  hipMemcpy(d_k,k,sizeof(fe),hipMemcpyHostToDevice); CUDA_CHECK_ERROR();
  mul_kernel<<<1,1>>>(d_r,d_p,*d_k,bits); CUDA_CHECK_ERROR();
  hipMemcpy(r,d_r,sizeof(pe),hipMemcpyDeviceToHost); CUDA_CHECK_ERROR();
  hipFree(d_r); hipFree(d_p); hipFree(d_k); CUDA_CHECK_ERROR();
}
